#include "hip/hip_runtime.h"
﻿struct GenerationData
{
	int Size;
	float distVertex;
	float xCell;
	float yCell;

	float Range;
	int Octaves;
	float Persistance;
	float Padding;
};

#define GRAD_X 80
#define GRAD_Y 40

struct Vector2
{
	float x;
	float y;
};

struct Vector3
{
	float x;
	float y;
	float z;
};

struct Vertex
{
	struct Vector3 VertexPosition;
	struct Vector2 TextureCoordinates;
	struct Vector3 VertexNormals;
	struct Vector3 Tangents;
	struct Vector3 BiNormals;
};

float Mag(struct Vector3 v)
{
	return sqrt((v.x*v.x) + (v.y*v.y) + (v.z*v.z));
};

void Normalize(struct Vector3* v)
{
	float mag = Mag(v[0]);
	v->x /= mag;
	v->y /= mag;
	v->z /= mag;
};

struct Vector3 Cross(struct Vector3 v1, struct Vector3 v2)
{
	struct Vector3 ret;
	ret.x = v1.y*v2.z - v1.z*v2.y;
	ret.y = v1.z*v2.x - v1.x*v2.z;
	ret.z = v1.x*v2.y - v1.y*v2.x;

	Normalize(&ret);

	return ret;
}

float Lerp(float a, float b, float t)
{
	return (a + (b - a)*t);
};

float Dot2(struct Vector2 v1, struct Vector2 v2)
{
	return v1.x * v2.x + v1.y * v2.y;
}

float Fade(float t)
{
	return t * t * t * (t * (t * 6 - 15) + 10);
};

float Noise(float x, float y, __global char* GradIndex, __global struct Vector2* GradientVector)
{
	int xi = x;
	xi %= GRAD_X;
	int yi = y;
	yi %= GRAD_Y;
	int xi2 = (xi + 1) % GRAD_X;
	int yi2 = (yi + 1) % GRAD_Y;


	struct Vector2 gTopLeft = GradientVector[GradIndex[xi*yi - 1]];
	struct Vector2 gTopRight = GradientVector[GradIndex[xi2*yi - 1]];
	struct Vector2 gBottomLeft = GradientVector[GradIndex[xi*yi2 - 1]];
	struct Vector2 gBottomRight = GradientVector[GradIndex[xi2*yi2 - 1]];

	float xf = x - floor(x);
	float yf = y - floor(y);
	float u = Fade(xf);
	float v = Fade(yf);

	xf *= 2.0f;
	yf *= 2.0f;
	xf -= 1.0f;
	yf -= 1.0f;

	struct Vector2 vTopLeft;
	vTopLeft.x = -1 - xf;
	vTopLeft.y = -1 - yf;

	struct Vector2 vTopRight;
	vTopRight.x = 1  - xf;
	vTopRight.y = -1 - yf;

	struct Vector2 vBottomLeft;
	vBottomLeft.x = -1 - xf;
	vBottomLeft.y = 1 - yf;

	struct Vector2 vBottomRight;
	vBottomRight.x = 1 - xf;
	vBottomRight.y = 1 - yf;

	return Lerp(
		
		Lerp(Dot2(vTopLeft, gTopLeft),
		Dot2(vTopRight, gTopRight),
		u)
		
		, 
		
		Lerp(Dot2(vBottomLeft, gBottomLeft),
			Dot2(vBottomRight, gBottomRight),
			u)
		
		, 
		
		v);
};

float OctaveNoise(float x, float y, int octaves, float persistance, __global char* GradIndex, __global struct Vector2* GradientVector)
{
	float total = 0;
	float frequency = 1;
	float amplitude = 1;
	float maxValue = 0;

	for (int i = 0; i < octaves; i++) {
		total += Noise(x * frequency, y * frequency, GradIndex, GradientVector) * amplitude;

		maxValue += amplitude;

		amplitude *= persistance;
		frequency *= 2;
	}

	return total / maxValue;
}

__kernel
void GenerateMesh(__global struct GenerationData* GenData, __global char* GradIndex, __global struct Vector2* GradientVector, __global struct Vertex* OutputVertex)
{
	int i = get_global_id(0);

	int x = i / GenData[0].Size;
	int y = i % GenData[0].Size;

	float xf = x * 10;
	float yf = y * 10;

	xf /= GenData[0].Size;
	yf /= GenData[0].Size;

	float res = OctaveNoise(xf + GenData[0].xCell, yf + GenData[0].yCell, GenData[0].Octaves, GenData[0].Persistance, GradIndex, GradientVector);


	OutputVertex[i].VertexPosition.x = GenData[0].distVertex*x;
	OutputVertex[i].VertexPosition.y = res*30;
	OutputVertex[i].VertexPosition.z = GenData[0].distVertex*y;

	OutputVertex[i].TextureCoordinates.x = x/4.0f;
	OutputVertex[i].TextureCoordinates.y = y/4.0f;
};

__kernel
void GenerateNormals(__global struct GenerationData* GenData, __global unsigned int*Indices, __global struct Vertex* OutputVertex)
{
	int i = get_global_id(0); // Lo que regresa es el indice de la operación que se está ejecutando.
	i = i * 3;

	unsigned int index1 = Indices[i];
	unsigned int index2 = Indices[i + 1];
	unsigned int index3 = Indices[i + 2];

	__global struct Vertex* Vert1 = &OutputVertex[index1];
	__global struct Vertex* Vert2 = &OutputVertex[index2];
	__global struct Vertex* Vert3 = &OutputVertex[index3];

	struct Vector3 V1;
	V1.x = Vert1->VertexPosition.x - Vert2->VertexPosition.x;
	V1.y = Vert1->VertexPosition.y - Vert2->VertexPosition.y;
	V1.z = Vert1->VertexPosition.z - Vert2->VertexPosition.z;

	struct Vector3 V2;
	V2.x = Vert1->VertexPosition.x - Vert3->VertexPosition.x;
	V2.y = Vert1->VertexPosition.y - Vert3->VertexPosition.y;
	V2.z = Vert1->VertexPosition.z - Vert3->VertexPosition.z;

	Normalize(&V1);
	Normalize(&V2);

	struct Vector3 normal = Cross(V1, V2);

	Normalize(&normal);

	OutputVertex[index1].VertexNormals = OutputVertex[index2].VertexNormals = OutputVertex[index3].VertexNormals = normal;
	OutputVertex[index1].Tangents = OutputVertex[index2].Tangents = OutputVertex[index3].Tangents = V1;
	OutputVertex[index1].BiNormals = OutputVertex[index2].BiNormals = OutputVertex[index3].BiNormals = V2;
};
