#include "hip/hip_runtime.h"
﻿struct GenerationData
{
	int Size;
	double distVertex;
	float xCell;
	float yCell;

	float Range;
	int Octaves;
	float Persistance;
	int CubeSide;

	float padding[3];
};

#define GRAD_X 250
#define GRAD_Y 50

struct Vector2
{
	float x;
	float y;
};

struct Vector3
{
	float x;
	float y;
	float z;
};

struct Vertex
{
	struct Vector3 VertexPosition;
	struct Vector2 TextureCoordinates;
	struct Vector3 VertexNormals;
	struct Vector3 Tangents;
	struct Vector3 BiNormals;
};

float Mag(struct Vector3 v)
{
	return sqrt((v.x*v.x) + (v.y*v.y) + (v.z*v.z));
};

void Normalize(struct Vector3* v)
{
	float mag = Mag(v[0]);
	v->x /= mag;
	v->y /= mag;
	v->z /= mag;
};

struct Vector3 Cross(struct Vector3 v1, struct Vector3 v2)
{
	struct Vector3 ret;
	ret.x = v1.y*v2.z - v1.z*v2.y;
	ret.y = v1.z*v2.x - v1.x*v2.z;
	ret.z = v1.x*v2.y - v1.y*v2.x;

	Normalize(&ret);

	return ret;
}

float Lerp(float a, float b, float t)
{
	return (a + (b - a)*t);
};

float Dot2(struct Vector2 v1, struct Vector2 v2)
{
	return v1.x * v2.x + v1.y * v2.y;
}

float Fade(float t)
{
	return t * t * t * (t * (t * 6 - 15) + 10);
};

float Noise(float x, float y, __global char* GradIndex, __global struct Vector2* GradientVector)
{
	int xi = x;
	xi %= GRAD_X;
	int yi = y;
	yi %= GRAD_Y;
	int xi2 = (xi + 1) % GRAD_X;
	int yi2 = (yi + 1) % GRAD_Y;


	struct Vector2 gTopLeft = GradientVector[GradIndex[xi*yi - 1]];
	struct Vector2 gTopRight = GradientVector[GradIndex[xi2*yi - 1]];
	struct Vector2 gBottomLeft = GradientVector[GradIndex[xi*yi2 - 1]];
	struct Vector2 gBottomRight = GradientVector[GradIndex[xi2*yi2 - 1]];

	float xf = x - floor(x);
	float yf = y - floor(y);
	float u = Fade(xf);
	float v = Fade(yf);

	xf *= 2.0f;
	yf *= 2.0f;
	xf -= 1.0f;
	yf -= 1.0f;

	struct Vector2 vTopLeft;
	vTopLeft.x = -1 - xf;
	vTopLeft.y = -1 - yf;

	struct Vector2 vTopRight;
	vTopRight.x = 1 - xf;
	vTopRight.y = -1 - yf;

	struct Vector2 vBottomLeft;
	vBottomLeft.x = -1 - xf;
	vBottomLeft.y = 1 - yf;

	struct Vector2 vBottomRight;
	vBottomRight.x = 1 - xf;
	vBottomRight.y = 1 - yf;

	return Lerp(

		Lerp(Dot2(vTopLeft, gTopLeft),
			Dot2(vTopRight, gTopRight),
			u)

		,

		Lerp(Dot2(vBottomLeft, gBottomLeft),
			Dot2(vBottomRight, gBottomRight),
			u)

		,

		v);
};

float OctaveNoise(float x, float y, int octaves, float persistance, __global char* GradIndex, __global struct Vector2* GradientVector)
{
	float total = 0;
	float frequency = 1;
	float amplitude = 1;
	float maxValue = 0;

	for (int i = 0; i < octaves; i++) 
	{
		total += Noise(x * frequency, y * frequency, GradIndex, GradientVector) * amplitude;
	
		maxValue += amplitude;
	
		amplitude *= persistance;
		frequency *= 2;
	}

	return total / maxValue;
}

void CubeToSphere(__global struct Vector3* cubepoint)
{
	float x2 = pow(cubepoint->x, 2);
	float y2 = pow(cubepoint->y, 2);
	float z2 = pow(cubepoint->z, 2);

	cubepoint->x = cubepoint->x * sqrt(1.0f - y2 / 2.0f -
		z2 / 2.0f +
		y2 * z2 / 3.0f);

	cubepoint->y = cubepoint->y * sqrt(1.0f - z2 / 2.0f -
		x2 / 2.0f +
		x2 * z2 / 3.0f);

	cubepoint->z = cubepoint->z * sqrt(1.0f - x2 / 2.0f -
		y2 / 2.0f +
		y2 * x2 / 3.0f);

}

__kernel
void GenerateMesh(__global struct GenerationData* GenData, __global char* GradIndex, __global struct Vector2* GradientVector, __global struct Vertex* OutputVertex)
{
	int i = get_global_id(0);

	int x = i / GenData[0].Size;
	int y = i % GenData[0].Size;

	float xf = x;
	float yf = y;

	xf /= GenData[0].Size;
	yf /= GenData[0].Size;

	double xxPos = (GenData[0].distVertex*x);
	double yxPos = (GenData[0].distVertex*y);

	double xPos = (xxPos / 512.0) / 50.0 + 1.0 / 50.0 * GenData[0].xCell;
	double yPos = (yxPos / 512.0) / 50.0 + 1.0 / 50.0 * GenData[0].yCell;

	OutputVertex[i].VertexPosition.x = xPos*2.0 - 1.0;
	OutputVertex[i].VertexPosition.y = yPos*2.0 - 1.0;
	OutputVertex[i].VertexPosition.z = -1.0;

	CubeToSphere(&OutputVertex[i].VertexPosition);

	OutputVertex[i].TextureCoordinates.x = xxPos/512.0f;
	OutputVertex[i].TextureCoordinates.y = yxPos/512.0f;
};

#define HEIGTHMAP_RES 1024

__kernel
void GenerateNoise(__global struct GenerationData* GenData, __global char* GradIndex, __global struct Vector2* GradientVector, __global float* OutputImage)
{
	int i = get_global_id(0);

	int y = i / HEIGTHMAP_RES;
	int x = i % HEIGTHMAP_RES;

	float xf = x + 0.5f;
	float yf = y + 0.5f;

	xf /= 1024.0f;
	yf /= 1024.0f;

	OutputImage[i] = (OctaveNoise(xf + GenData[0].xCell, yf + GenData[0].yCell, GenData[0].Octaves, GenData[0].Persistance, GradIndex, GradientVector));
};

__kernel
void GenerateNormalTexture(__global float* NoiseTexture, __global struct Vector3* OutNormalTexture)
{
	/*int i = get_global_id(0);

	int y = i / HEIGTHMAP_RES;
	int x = i % HEIGTHMAP_RES; 

	float xf = x + 0.5f;
	float yf = y + 0.5f;

	xf /= 1024.0f;
	yf /= 1024.0f;

	OutputImage[i] = (OctaveNoise(xf + GenData[0].xCell, yf + GenData[0].yCell, GenData[0].Octaves, GenData[0].Persistance, GradIndex, GradientVector));*/
};

__kernel
void GenerateNormals(__global struct GenerationData* GenData, __global unsigned int*Indices, __global struct Vertex* OutputVertex)
{
	int i = get_global_id(0); // Lo que regresa es el indice de la operación que se está ejecutando.
	i = i * 3;

	unsigned int index1 = Indices[i];
	unsigned int index2 = Indices[i + 1];
	unsigned int index3 = Indices[i + 2];

	__global struct Vertex* Vert1 = &OutputVertex[index1];
	__global struct Vertex* Vert2 = &OutputVertex[index2];
	__global struct Vertex* Vert3 = &OutputVertex[index3];

	struct Vector3 V1;
	V1.x = Vert1->VertexPosition.x - Vert2->VertexPosition.x;
	V1.y = Vert1->VertexPosition.y - Vert2->VertexPosition.y;
	V1.z = Vert1->VertexPosition.z - Vert2->VertexPosition.z;

	struct Vector3 V2;
	V2.x = Vert1->VertexPosition.x - Vert3->VertexPosition.x;
	V2.y = Vert1->VertexPosition.y - Vert3->VertexPosition.y;
	V2.z = Vert1->VertexPosition.z - Vert3->VertexPosition.z;

	Normalize(&V1);
	Normalize(&V2);

	struct Vector3 normal = Cross(V1, V2);

	Normalize(&normal);

	OutputVertex[index1].VertexNormals = OutputVertex[index2].VertexNormals = OutputVertex[index3].VertexNormals = normal;
	OutputVertex[index1].Tangents = OutputVertex[index2].Tangents = OutputVertex[index3].Tangents = V1;
	OutputVertex[index1].BiNormals = OutputVertex[index2].BiNormals = OutputVertex[index3].BiNormals = V2;
};
