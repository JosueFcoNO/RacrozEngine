﻿__kernel
void sum(__global int* OriginalNumber, __global int*NewNumber)
{
	int i = get_global_id(0); // Lo que regresa es el indice de la operación que se está ejecutando.

	NewNumber[i] = OriginalNumber[i] * OriginalNumber[i];
}
